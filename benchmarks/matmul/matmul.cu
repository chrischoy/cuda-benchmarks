#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdlib>
#include <hipcub/hipcub.hpp>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <torch/extension.h>
#include <vector>

#include "../../include/vector_types.cuh"
#include "matmul_common.cuh"

using namespace cuda_benchmark::matmul;
using namespace nvcuda;
#if 1
// Forward declarations to link to wrappers in matmul_wmma_sm80.cu
extern "C" {
void launch_wmma_db_ampere_f16_f32(dim3, dim3, hipStream_t, const half *,
                                   const half *, const half *, const long *,
                                   const long *, float *, int, int, int, int,
                                   int);
void launch_wmma_db_ampere_f16_f16(dim3, dim3, hipStream_t, const half *,
                                   const half *, const half *, const long *,
                                   const long *, half *, int, int, int, int,
                                   int);
// Non-atomic store variants
void launch_wmma_db_ampere_f16_f32_store(dim3, dim3, hipStream_t, const half *,
                                         const half *, const half *,
                                         const long *, const long *, float *,
                                         int, int, int, int, int);
void launch_wmma_db_ampere_f16_f16_store(dim3, dim3, hipStream_t, const half *,
                                         const half *, const half *,
                                         const long *, const long *, half *,
                                         int, int, int, int, int);
}
#endif

// Compute D[d_inds, :] += A[a_inds, :] @ B + C[a_inds, :]

// Naive FP32 baseline: one block per output row p, threads iterate N
__global__ void implicit_gemm_naive_f32(const float *__restrict__ A, // [M, K]
                                        const float *__restrict__ B, // [K, N]
                                        const float *__restrict__ C, // [M, N]
                                        const long *__restrict__ a_inds, // [P]
                                        const long *__restrict__ d_inds, // [P]
                                        float *__restrict__ D, // [Q, N]
                                        int M, int K, int N, int P, int Q) {
  for (int p = blockIdx.y; p < P; p += gridDim.y) {
    int n0 = blockIdx.x * blockDim.x + threadIdx.x;
    long a = a_inds[p];
    long d = d_inds[p];
    if (a < 0 || a >= M || d < 0 || d >= Q)
      continue;

    for (int n = n0; n < N; n += blockDim.x * gridDim.x) {
      float acc = 0.0f;
      const float *a_row = A + a * K;
      const float *b_col = B + n; // column-major access simulated via stride N
      for (int k = 0; k < K; ++k) {
        acc += a_row[k] * b_col[k * N];
      }
      float c_val = C ? C[a * N + n] : 0.0f;
      atomicAdd(&D[d * N + n], acc + c_val);
    }
  }
}

// WMMA FP16 inputs, FP32 accumulate, 16x16x16 tiles; gather/scatter per tile
__global__ void implicit_gemm_wmma_f16_acc_f32(
    const half *__restrict__ A,      // [M, K]
    const half *__restrict__ B,      // [K, N]
    const half *__restrict__ C,      // [M, N] optional, can be null
    const long *__restrict__ a_inds, // [P]
    const long *__restrict__ d_inds, // [P]
    float *__restrict__ D,           // [Q, N]
    int M, int K, int N, int P, int Q) {
  // Tile indices
  int tile_n = blockIdx.x; // along N
  for (int tile_p = blockIdx.y; tile_p < (P + 15) / 16; tile_p += gridDim.y) {
    int warp_lane = threadIdx.x % 32;

    // Load up to 16 a/d indices participating in this tile row
    __shared__ long a_row_idx[16];
    __shared__ long d_row_idx[16];
    int row_in_tile = warp_lane;
    if (row_in_tile < 16) {
      int p = tile_p * 16 + row_in_tile;
      long aval = (p < P) ? a_inds[p] : -1;
      long dval = (p < P) ? d_inds[p] : -1;
      a_row_idx[row_in_tile] = aval;
      d_row_idx[row_in_tile] = dval;
    }
    __syncthreads();

    // Shared tiles and WMMA fragments
    __shared__ half A_tile[16 * 16];
    __shared__ half B_tile[16 * 16];
    __shared__ float out_tile[16 * 16];

    // WMMA fragments
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;
    wmma::fill_fragment(c_frag, 0.0f);

    int n_base = tile_n * 16;
    // Iterate over K in 16 chunks
    for (int k0 = 0; k0 < K; k0 += 16) {
      // Each lane cooperatively loads 256 elements in 8 iterations
      for (int t = threadIdx.x % 32; t < 256; t += 32) {
        int r = t / 16;
        int c = t % 16;
        long arow = a_row_idx[r];
        half av = __float2half(0.0f);
        if (arow >= 0 && arow < M && (k0 + c) < K) {
          av = A[arow * K + (k0 + c)];
        }
        A_tile[r * 16 + c] = av;

        // For B_tile column-major layout expected by WMMA, write at (c*ld + r)
        half bv = __float2half(0.0f);
        int kb = k0 + r;
        int nb = n_base + c;
        if (kb < K && nb < N) {
          bv = B[kb * N + nb];
        }
        B_tile[c * 16 + r] = bv;
      }
      __syncthreads();

      // Load fragments
      wmma::load_matrix_sync(a_frag, A_tile, 16);
      wmma::load_matrix_sync(b_frag, B_tile, 16);
      wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
      __syncthreads();
    }

    // Epilogue: add optional bias C[a_inds,:] and scatter-add to D[d_inds,:]
    // Store accumulator to smem then atomically add
    wmma::store_matrix_sync(out_tile, c_frag, 16, wmma::mem_row_major);
    __syncthreads();

    // Each lane writes 8 elements via atomicAdd
    for (int t = threadIdx.x % 32; t < 256; t += 32) {
      int r = t / 16;
      int c = t % 16;
      int n = n_base + c;
      int p = tile_p * 16 + r;
      long arow = a_row_idx[r];
      long drow = d_row_idx[r];
      if (p < P && arow >= 0 && arow < M && drow >= 0 && drow < Q && n < N) {
        float val = out_tile[r * 16 + c];
        if (C) {
          val += __half2float(C[arow * N + n]);
        }
        atomicAdd(&D[drow * N + n], val);
      }
    }
  }
}

#if 1
// ---- Ampere cp.async helpers (16B at a time) --------------------------------
static __device__ __forceinline__ void
cp_async_16B(void *smem_ptr, const void *gmem_ptr, bool pred) {
#if __CUDA_ARCH__ >= 800
  if (pred) {
    unsigned smem_addr =
        static_cast<unsigned>(__cvta_generic_to_shared(smem_ptr));
    asm volatile("cp.async.cg.shared.global [%0], [%1], %2;\n" ::"r"(smem_addr),
                 "l"(gmem_ptr), "n"(16));
  } else {
    uint4 z = {0, 0, 0, 0};
    *reinterpret_cast<uint4 *>(smem_ptr) = z;
  }
#else
  if (pred) {
    uint4 v = *reinterpret_cast<const uint4 *>(gmem_ptr);
    *reinterpret_cast<uint4 *>(smem_ptr) = v;
  } else {
    uint4 z = {0, 0, 0, 0};
    *reinterpret_cast<uint4 *>(smem_ptr) = z;
  }
#endif
}

static __device__ __forceinline__ void cp_async_commit() {
#if __CUDA_ARCH__ >= 800
  asm volatile("cp.async.commit_group;\n");
#endif
}
static __device__ __forceinline__ void cp_async_wait_all() {
#if __CUDA_ARCH__ >= 800
  asm volatile("cp.async.wait_group 0;\n");
#endif
}

// ---- Kernel: WMMA f16×f16→f32, 16x16 tiles, double-buffered over K ----------
// Assumes blockDim.x == 32 (single warp per tile).
__global__ void implicit_gemm_wmma_f16_acc_f32_db_ampere(
    const half *__restrict__ A,      // [M, K], row-major
    const half *__restrict__ B,      // [K, N], row-major
    const half *__restrict__ Cbias,  // [M, N] optional (may be nullptr)
    const long *__restrict__ a_inds, // [P] (rows to gather from A/C)
    const long *__restrict__ d_inds, // [P] (rows to scatter-add into D)
    float *__restrict__ D,           // [Q, N]
    int M, int K, int N, int P, int Q) {
  const int lane = threadIdx.x & 31;
  const int tile_n = blockIdx.x;
  const int n_base = tile_n * 16;
  for (int tile_p = blockIdx.y; tile_p < (P + 15) / 16; tile_p += gridDim.y) {

    __shared__ long a_row_idx[16];
    __shared__ long d_row_idx[16];
    if (lane < 16) {
      const int p = tile_p * 16 + lane;
      a_row_idx[lane] = (p < P) ? a_inds[p] : -1;
      d_row_idx[lane] = (p < P) ? d_inds[p] : -1;
    }
    __syncthreads();

    __shared__ __align__(16) half Asmem[2][16 * 16];
    __shared__ __align__(16) half Bsmem[2][16 * 16];
    __shared__ float OutTile[16 * 16];

    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> acc_frag;
    wmma::fill_fragment(acc_frag, 0.0f);

    // Helper to stage tiles for a given k0 into buffer `stage`
    auto stage_load_tiles = [&](int k0, int stage_param) {
      const int row = lane & 15;
      const int seg = lane >> 4;

      long arow = a_row_idx[row];
      const half *gA = (arow >= 0 && arow < M)
                           ? (A + arow * (size_t)K + k0 + seg * 8)
                           : nullptr;
      bool predA = (arow >= 0 && arow < M) && (k0 + seg * 8 + 7 < K);
      half *sA = &Asmem[stage_param][row * 16 + seg * 8];
      cp_async_16B((void *)sA, (const void *)gA, predA);

      const int kb = k0 + row;
      const int nb = n_base + seg * 8;
      const half *gB = (kb < K && nb < N) ? (B + (size_t)kb * N + nb) : nullptr;
      bool predB = (kb < K) && (nb + 7 < N);
      half *sB = &Bsmem[stage_param][row * 16 + seg * 8];
      cp_async_16B((void *)sB, (const void *)gB, predB);
    };

    int stage = 0;
    stage_load_tiles(/*k0=*/0, /*stage=*/stage);
    cp_async_commit();
    cp_async_wait_all();
    __syncthreads();

    for (int k0 = 0; k0 < K; k0 += 16) {
      const int next_stage = stage ^ 1;
      if (k0 + 16 < K) {
        stage_load_tiles(k0 + 16, next_stage);
        cp_async_commit();
      }

      wmma::load_matrix_sync(a_frag, &Asmem[stage][0], 16);
      wmma::load_matrix_sync(b_frag, &Bsmem[stage][0], 16);
      wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);

      if (k0 + 16 < K) {
        cp_async_wait_all();
      }
      __syncthreads();
      stage = next_stage;
    }

    wmma::store_matrix_sync(OutTile, acc_frag, 16, wmma::mem_row_major);
    __syncthreads();

    for (int t = lane; t < 256; t += 32) {
      int r = t / 16;
      int c = t % 16;
      int n = n_base + c;
      int p = tile_p * 16 + r;
      long arow = a_row_idx[r];
      long drow = d_row_idx[r];

      if (p < P && arow >= 0 && arow < M && drow >= 0 && drow < Q && n < N) {
        float val = OutTile[r * 16 + c];
        if (Cbias) {
          val += __half2float(Cbias[(size_t)arow * N + n]);
        }
        atomicAdd(&D[(size_t)drow * N + n], val);
      }
    }
  }
}
#endif

#if 0
// BF16 WMMA kernel intentionally disabled
__global__ void implicit_gemm_wmma_bf16_acc_f32(
    const __hip_bfloat16* __restrict__ A,
    const __hip_bfloat16* __restrict__ B,
    const __hip_bfloat16* __restrict__ C,
    const long* __restrict__ a_inds,
    const long* __restrict__ d_inds,
    float* __restrict__ D,
    int M, int K, int N, int P, int Q) {}
#endif

// Inline-PTX assisted FP32 kernel: each thread computes 4 contiguous columns
// using ld.global.v4.f32 for B and C
__global__ void
implicit_gemm_f32_ptx_v4(const float *__restrict__ A,     // [M, K]
                         const float *__restrict__ B,     // [K, N]
                         const float *__restrict__ C,     // [M, N] optional
                         const long *__restrict__ a_inds, // [P]
                         const long *__restrict__ d_inds, // [P]
                         float *__restrict__ D,           // [Q, N]
                         int M, int K, int N, int P, int Q) {
  int thread_group_cols = 4;
  int n0 = (blockIdx.x * blockDim.x + threadIdx.x) * thread_group_cols;
  if (n0 >= N)
    return;

  for (int p = blockIdx.y; p < P; p += gridDim.y) {
    long a = a_inds[p];
    long d = d_inds[p];
    if (a < 0 || a >= M || d < 0 || d >= Q)
      continue;

    float acc0 = 0.0f, acc1 = 0.0f, acc2 = 0.0f, acc3 = 0.0f;
    const float *a_row = A + a * K;

    // Iterate K and accumulate 4 outputs (columns n0..n0+3)
    for (int k = 0; k < K; ++k) {
      float a_val = a_row[k];
      const float *b_ptr = B + k * N + n0;
      float b0, b1, b2, b3;
      bool in_bounds4 = (n0 + 3) < N;
      if (in_bounds4 && ((reinterpret_cast<uintptr_t>(b_ptr) & 0xF) == 0)) {
        asm volatile("{\n\t"
                     "ld.global.v4.f32 {%0, %1, %2, %3}, [%4];\n\t"
                     "}\n"
                     : "=f"(b0), "=f"(b1), "=f"(b2), "=f"(b3)
                     : "l"(b_ptr));
      } else {
        b0 = (n0 + 0) < N ? b_ptr[0] : 0.0f;
        b1 = (n0 + 1) < N ? b_ptr[1] : 0.0f;
        b2 = (n0 + 2) < N ? b_ptr[2] : 0.0f;
        b3 = (n0 + 3) < N ? b_ptr[3] : 0.0f;
      }
      acc0 += a_val * b0;
      acc1 += a_val * b1;
      acc2 += a_val * b2;
      acc3 += a_val * b3;
    }

    if (C) {
      const float *c_ptr = C + a * N + n0;
      float c0, c1, c2, c3;
      bool in_bounds4 = (n0 + 3) < N;
      if (in_bounds4 && ((reinterpret_cast<uintptr_t>(c_ptr) & 0xF) == 0)) {
        asm volatile("{\n\t"
                     "ld.global.v4.f32 {%0, %1, %2, %3}, [%4];\n\t"
                     "}\n"
                     : "=f"(c0), "=f"(c1), "=f"(c2), "=f"(c3)
                     : "l"(c_ptr));
      } else {
        c0 = (n0 + 0) < N ? c_ptr[0] : 0.0f;
        c1 = (n0 + 1) < N ? c_ptr[1] : 0.0f;
        c2 = (n0 + 2) < N ? c_ptr[2] : 0.0f;
        c3 = (n0 + 3) < N ? c_ptr[3] : 0.0f;
      }
      acc0 += c0;
      acc1 += c1;
      acc2 += c2;
      acc3 += c3;
    }

    float *d_ptr = D + d * N + n0;
    if ((n0 + 0) < N)
      atomicAdd(&d_ptr[0], acc0);
    if ((n0 + 1) < N)
      atomicAdd(&d_ptr[1], acc1);
    if ((n0 + 2) < N)
      atomicAdd(&d_ptr[2], acc2);
    if ((n0 + 3) < N)
      atomicAdd(&d_ptr[3], acc3);
  }
}

// ---- CUB-based gather-scatter matmul (templated datatypes)
// -------------------
namespace {

template <typename T> struct ToFloat;

template <> struct ToFloat<float> {
  static __device__ __forceinline__ float convert(float x) { return x; }
};

template <> struct ToFloat<half> {
  static __device__ __forceinline__ float convert(half x) {
    return __half2float(x);
  }
};

template <> struct ToFloat<__hip_bfloat16> {
  static __device__ __forceinline__ float convert(__hip_bfloat16 x) {
    return __bfloat162float(x);
  }
};

template <typename T> struct FromFloat;

template <> struct FromFloat<float> {
  static __device__ __forceinline__ float convert(float x) { return x; }
};

template <> struct FromFloat<half> {
  static __device__ __forceinline__ half convert(float x) {
    return __float2half(x);
  }
};

template <> struct FromFloat<__hip_bfloat16> {
  static __device__ __forceinline__ __hip_bfloat16 convert(float x) {
    return __float2bfloat16(x);
  }
};

// Atomic add for float/half/bfloat16
static __device__ __forceinline__ void atomicAddTyped(float *addr, float val) {
  atomicAdd(addr, val);
}

static __device__ __forceinline__ void atomicAddTyped(half *addr, float val) {
  // Implement via CAS on 32-bit word containing the target half
  uintptr_t int_addr = reinterpret_cast<uintptr_t>(addr);
  unsigned int *base = reinterpret_cast<unsigned int *>(int_addr & ~0x3ULL);
  bool high = (int_addr & 0x2ULL) != 0ULL;
  unsigned int old = *base;
  unsigned int assumed;
  do {
    assumed = old;
    unsigned short hbits = high
                               ? static_cast<unsigned short>(assumed >> 16)
                               : static_cast<unsigned short>(assumed & 0xFFFFu);
    half hval = __ushort_as_half(hbits);
    float f = __half2float(hval) + val;
    unsigned short new_hbits = __half_as_ushort(__float2half(f));
    unsigned int new_word =
        high ? ((assumed & 0x0000FFFFu) |
                (static_cast<unsigned int>(new_hbits) << 16))
             : ((assumed & 0xFFFF0000u) | static_cast<unsigned int>(new_hbits));
    old = atomicCAS(base, assumed, new_word);
  } while (old != assumed);
}

static __device__ __forceinline__ void atomicAddTyped(__hip_bfloat16 *addr,
                                                      float val) {
  uintptr_t int_addr = reinterpret_cast<uintptr_t>(addr);
  unsigned int *base = reinterpret_cast<unsigned int *>(int_addr & ~0x3ULL);
  bool high = (int_addr & 0x2ULL) != 0ULL;
  unsigned int old = *base;
  unsigned int assumed;
  do {
    assumed = old;
    unsigned short bbits = high
                               ? static_cast<unsigned short>(assumed >> 16)
                               : static_cast<unsigned short>(assumed & 0xFFFFu);
    __hip_bfloat16 bval = __ushort_as_bfloat16(bbits);
    float f = __bfloat162float(bval) + val;
    unsigned short new_bbits = __bfloat16_as_ushort(__float2bfloat16(f));
    unsigned int new_word =
        high ? ((assumed & 0x0000FFFFu) |
                (static_cast<unsigned int>(new_bbits) << 16))
             : ((assumed & 0xFFFF0000u) | static_cast<unsigned int>(new_bbits));
    old = atomicCAS(base, assumed, new_word);
  } while (old != assumed);
}

template <typename TA, typename TB, typename TC, typename TD, int BLOCK_THREADS,
          int ITEMS_PER_THREAD>
__global__ void
implicit_gemm_cub_blockload(const TA *__restrict__ A,    // [M, K]
                            const TB *__restrict__ B,    // [K, N]
                            const TC *__restrict__ Copt, // [M, N] or nullptr
                            const long *__restrict__ a_inds, // [P]
                            const long *__restrict__ d_inds, // [P]
                            TD *__restrict__ D,              // [Q, N]
                            int M, int K, int N, int P, int Q) {
  constexpr int TILE = BLOCK_THREADS * ITEMS_PER_THREAD;
  int n0 = blockIdx.x * TILE;
  int tid = threadIdx.x;

  using BlockLoadB = hipcub::BlockLoad<TB, BLOCK_THREADS, ITEMS_PER_THREAD,
                                    hipcub::BLOCK_LOAD_VECTORIZE>;
  using BlockLoadC = hipcub::BlockLoad<TC, BLOCK_THREADS, ITEMS_PER_THREAD,
                                    hipcub::BLOCK_LOAD_VECTORIZE>;
  __shared__ typename BlockLoadB::TempStorage temp_storage_b;
  __shared__ typename BlockLoadC::TempStorage temp_storage_c;
  for (int p = blockIdx.y; p < P; p += gridDim.y) {
    long a = a_inds[p];
    long d = d_inds[p];
    if (a < 0 || a >= M || d < 0 || d >= Q)
      continue;

    float acc[ITEMS_PER_THREAD];
#pragma unroll
    for (int i = 0; i < ITEMS_PER_THREAD; ++i)
      acc[i] = 0.0f;

    // Iterate over K
    for (int k = 0; k < K; ++k) {
      float aval = ToFloat<TA>::convert(A[a * (size_t)K + k]);

      // Load a TILE segment from B[k, n0:n0+TILE)
      TB b_vals[ITEMS_PER_THREAD];
      int valid = 0;
      if (n0 < N) {
        int remaining = N - n0;
        valid = remaining > TILE ? TILE : remaining;
      }
      BlockLoadB(temp_storage_b)
          .Load(B + (size_t)k * N + n0, b_vals, valid, TB());
      __syncthreads();

#pragma unroll
      for (int i = 0; i < ITEMS_PER_THREAD; ++i) {
        acc[i] += aval * ToFloat<TB>::convert(b_vals[i]);
      }
    }

    // Optional bias C[a, n]
    if (Copt) {
      TC c_vals[ITEMS_PER_THREAD];
      int valid = 0;
      if (n0 < N) {
        int remaining = N - n0;
        valid = remaining > TILE ? TILE : remaining;
      }
      BlockLoadC(temp_storage_c)
          .Load(Copt + (size_t)a * N + n0, c_vals, valid, TC());
      __syncthreads();
#pragma unroll
      for (int i = 0; i < ITEMS_PER_THREAD; ++i) {
        acc[i] += ToFloat<TC>::convert(c_vals[i]);
      }
    }

    // Scatter-add into D[d, n]
    int base_n = n0 + tid * ITEMS_PER_THREAD;
#pragma unroll
    for (int i = 0; i < ITEMS_PER_THREAD; ++i) {
      int n = base_n + i;
      if (n < N) {
        atomicAddTyped(D + (size_t)d * N + n, acc[i]);
      }
    }
  }
}

} // anonymous namespace

// Host entry
std::vector<float>
benchmark_implicit_gemm(torch::Tensor A, torch::Tensor B, torch::Tensor C,
                        torch::Tensor a_inds, torch::Tensor d_inds,
                        torch::Tensor D, int method, int iterations = 100,
                        bool use_store = false) {
  int M = A.size(0);
  int K = A.size(1);
  int N = B.size(1);
  int P = a_inds.size(0);
  int Q = D.size(0);

  // Move to GPU/contig
  A = A.cuda().contiguous();
  B = B.cuda().contiguous();
  if (C.defined() && C.numel() > 0)
    C = C.cuda().contiguous();
  a_inds = a_inds.cuda().contiguous();
  d_inds = d_inds.cuda().contiguous();
  D = D.cuda().contiguous();

  dim3 block = calculate_block_size(method);
  dim3 grid = calculate_grid_size(P, N, method, block);

  // Warmup
  for (int i = 0; i < 5; ++i) {
    if (method == NAIVE_F32) {
      implicit_gemm_naive_f32<<<grid, block>>>(
          A.data_ptr<float>(), B.data_ptr<float>(),
          C.defined() ? C.data_ptr<float>() : nullptr, a_inds.data_ptr<long>(),
          d_inds.data_ptr<long>(), D.data_ptr<float>(), M, K, N, P, Q);
    } else if (method == WMMA_F16_ACC_F32) {
      implicit_gemm_wmma_f16_acc_f32<<<grid, dim3(32, 1, 1)>>>(
          reinterpret_cast<half *>(A.data_ptr<at::Half>()),
          reinterpret_cast<half *>(B.data_ptr<at::Half>()),
          C.defined() ? reinterpret_cast<half *>(C.data_ptr<at::Half>())
                      : nullptr,
          a_inds.data_ptr<long>(), d_inds.data_ptr<long>(), D.data_ptr<float>(),
          M, K, N, P, Q);
    } else if (method == F32_PTX_V4) {
      implicit_gemm_f32_ptx_v4<<<grid, block>>>(
          A.data_ptr<float>(), B.data_ptr<float>(),
          C.defined() ? C.data_ptr<float>() : nullptr, a_inds.data_ptr<long>(),
          d_inds.data_ptr<long>(), D.data_ptr<float>(), M, K, N, P, Q);
    } else if (method == WMMA_F16_ACC_F32_DB_AMPERE) {
      implicit_gemm_wmma_f16_acc_f32_db_ampere<<<grid, dim3(32, 1, 1)>>>(
          reinterpret_cast<half *>(A.data_ptr<at::Half>()),
          reinterpret_cast<half *>(B.data_ptr<at::Half>()),
          C.defined() ? reinterpret_cast<half *>(C.data_ptr<at::Half>())
                      : nullptr,
          a_inds.data_ptr<long>(), d_inds.data_ptr<long>(), D.data_ptr<float>(),
          M, K, N, P, Q);
    } else if (method == WMMA_DB_AMPERE_GENERIC ||
               method == WMMA_DB_AMPERE_GENERIC_STORE) {
      // Use SM80 WMMA double-buffer kernel; optional non-atomic store via
      // use_store flag
      if (method == WMMA_DB_AMPERE_GENERIC_STORE)
        use_store = true;
      auto Adtype = A.scalar_type();
      auto Ddtype = D.scalar_type();
      if (Adtype == at::kHalf && Ddtype == at::kFloat) {
        if (use_store) {
          launch_wmma_db_ampere_f16_f32_store(
              grid, dim3(32, 1, 1), nullptr,
              reinterpret_cast<half *>(A.data_ptr<at::Half>()),
              reinterpret_cast<half *>(B.data_ptr<at::Half>()),
              C.defined() ? reinterpret_cast<half *>(C.data_ptr<at::Half>())
                          : nullptr,
              a_inds.data_ptr<long>(), d_inds.data_ptr<long>(),
              D.data_ptr<float>(), M, K, N, P, Q);
        } else {
          launch_wmma_db_ampere_f16_f32(
              grid, dim3(32, 1, 1), nullptr,
              reinterpret_cast<half *>(A.data_ptr<at::Half>()),
              reinterpret_cast<half *>(B.data_ptr<at::Half>()),
              C.defined() ? reinterpret_cast<half *>(C.data_ptr<at::Half>())
                          : nullptr,
              a_inds.data_ptr<long>(), d_inds.data_ptr<long>(),
              D.data_ptr<float>(), M, K, N, P, Q);
        }
      } else if (Adtype == at::kHalf && Ddtype == at::kHalf) {
        if (use_store) {
          launch_wmma_db_ampere_f16_f16_store(
              grid, dim3(32, 1, 1), nullptr,
              reinterpret_cast<half *>(A.data_ptr<at::Half>()),
              reinterpret_cast<half *>(B.data_ptr<at::Half>()),
              C.defined() ? reinterpret_cast<half *>(C.data_ptr<at::Half>())
                          : nullptr,
              a_inds.data_ptr<long>(), d_inds.data_ptr<long>(),
              reinterpret_cast<half *>(D.data_ptr<at::Half>()), M, K, N, P, Q);
        } else {
          launch_wmma_db_ampere_f16_f16(
              grid, dim3(32, 1, 1), nullptr,
              reinterpret_cast<half *>(A.data_ptr<at::Half>()),
              reinterpret_cast<half *>(B.data_ptr<at::Half>()),
              C.defined() ? reinterpret_cast<half *>(C.data_ptr<at::Half>())
                          : nullptr,
              a_inds.data_ptr<long>(), d_inds.data_ptr<long>(),
              reinterpret_cast<half *>(D.data_ptr<at::Half>()), M, K, N, P, Q);
        }
      }
    } else if (method == CUB_F32_BLOCKLOAD) {
      auto Adtype = A.scalar_type();
      auto Bdtype = B.scalar_type();
      auto Ddtype = D.scalar_type();
      // Support: A/B/C in half or bfloat16; D in float/half/bfloat16
      if (Adtype == at::kHalf && Bdtype == at::kHalf) {
        if (Ddtype == at::kFloat) {
          implicit_gemm_cub_blockload<half, half, half, float, 128, 4>
              <<<grid, dim3(128, 1, 1)>>>(
                  reinterpret_cast<half *>(A.data_ptr<at::Half>()),
                  reinterpret_cast<half *>(B.data_ptr<at::Half>()),
                  C.defined() ? reinterpret_cast<half *>(C.data_ptr<at::Half>())
                              : nullptr,
                  a_inds.data_ptr<long>(), d_inds.data_ptr<long>(),
                  D.data_ptr<float>(), M, K, N, P, Q);
        } else if (Ddtype == at::kHalf) {
          implicit_gemm_cub_blockload<half, half, half, half, 128, 4>
              <<<grid, dim3(128, 1, 1)>>>(
                  reinterpret_cast<half *>(A.data_ptr<at::Half>()),
                  reinterpret_cast<half *>(B.data_ptr<at::Half>()),
                  C.defined() ? reinterpret_cast<half *>(C.data_ptr<at::Half>())
                              : nullptr,
                  a_inds.data_ptr<long>(), d_inds.data_ptr<long>(),
                  reinterpret_cast<half *>(D.data_ptr<at::Half>()), M, K, N, P,
                  Q);
        } else if (Ddtype == at::kBFloat16) {
          implicit_gemm_cub_blockload<half, half, half, __hip_bfloat16, 128, 4>
              <<<grid, dim3(128, 1, 1)>>>(
                  reinterpret_cast<half *>(A.data_ptr<at::Half>()),
                  reinterpret_cast<half *>(B.data_ptr<at::Half>()),
                  C.defined() ? reinterpret_cast<half *>(C.data_ptr<at::Half>())
                              : nullptr,
                  a_inds.data_ptr<long>(), d_inds.data_ptr<long>(),
                  reinterpret_cast<__hip_bfloat16 *>(D.data_ptr<at::BFloat16>()),
                  M, K, N, P, Q);
        }
      } else if (Adtype == at::kBFloat16 && Bdtype == at::kBFloat16) {
        if (Ddtype == at::kFloat) {
          implicit_gemm_cub_blockload<__hip_bfloat16, __hip_bfloat16,
                                      __hip_bfloat16, float, 128, 4>
              <<<grid, dim3(128, 1, 1)>>>(
                  reinterpret_cast<__hip_bfloat16 *>(A.data_ptr<at::BFloat16>()),
                  reinterpret_cast<__hip_bfloat16 *>(B.data_ptr<at::BFloat16>()),
                  C.defined() ? reinterpret_cast<__hip_bfloat16 *>(
                                    C.data_ptr<at::BFloat16>())
                              : nullptr,
                  a_inds.data_ptr<long>(), d_inds.data_ptr<long>(),
                  D.data_ptr<float>(), M, K, N, P, Q);
        } else if (Ddtype == at::kBFloat16) {
          implicit_gemm_cub_blockload<__hip_bfloat16, __hip_bfloat16,
                                      __hip_bfloat16, __hip_bfloat16, 128, 4>
              <<<grid, dim3(128, 1, 1)>>>(
                  reinterpret_cast<__hip_bfloat16 *>(A.data_ptr<at::BFloat16>()),
                  reinterpret_cast<__hip_bfloat16 *>(B.data_ptr<at::BFloat16>()),
                  C.defined() ? reinterpret_cast<__hip_bfloat16 *>(
                                    C.data_ptr<at::BFloat16>())
                              : nullptr,
                  a_inds.data_ptr<long>(), d_inds.data_ptr<long>(),
                  reinterpret_cast<__hip_bfloat16 *>(D.data_ptr<at::BFloat16>()),
                  M, K, N, P, Q);
        } else if (Ddtype == at::kHalf) {
          implicit_gemm_cub_blockload<__hip_bfloat16, __hip_bfloat16,
                                      __hip_bfloat16, half, 128, 4>
              <<<grid, dim3(128, 1, 1)>>>(
                  reinterpret_cast<__hip_bfloat16 *>(A.data_ptr<at::BFloat16>()),
                  reinterpret_cast<__hip_bfloat16 *>(B.data_ptr<at::BFloat16>()),
                  C.defined() ? reinterpret_cast<__hip_bfloat16 *>(
                                    C.data_ptr<at::BFloat16>())
                              : nullptr,
                  a_inds.data_ptr<long>(), d_inds.data_ptr<long>(),
                  reinterpret_cast<half *>(D.data_ptr<at::Half>()), M, K, N, P,
                  Q);
        }
      }
    }
  }
  hipDeviceSynchronize();
  // Reset D to avoid warmup accumulation affecting correctness/timing
  D.zero_();
  hipDeviceSynchronize();

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  std::vector<float> times;
  times.reserve(iterations);

  for (int it = 0; it < iterations; ++it) {
    hipEventRecord(start);
    if (method == NAIVE_F32) {
      implicit_gemm_naive_f32<<<grid, block>>>(
          A.data_ptr<float>(), B.data_ptr<float>(),
          C.defined() ? C.data_ptr<float>() : nullptr, a_inds.data_ptr<long>(),
          d_inds.data_ptr<long>(), D.data_ptr<float>(), M, K, N, P, Q);
    } else if (method == WMMA_F16_ACC_F32) {
      implicit_gemm_wmma_f16_acc_f32<<<grid, dim3(32, 1, 1)>>>(
          reinterpret_cast<half *>(A.data_ptr<at::Half>()),
          reinterpret_cast<half *>(B.data_ptr<at::Half>()),
          C.defined() ? reinterpret_cast<half *>(C.data_ptr<at::Half>())
                      : nullptr,
          a_inds.data_ptr<long>(), d_inds.data_ptr<long>(), D.data_ptr<float>(),
          M, K, N, P, Q);
    } else if (method == F32_PTX_V4) {
      implicit_gemm_f32_ptx_v4<<<grid, block>>>(
          A.data_ptr<float>(), B.data_ptr<float>(),
          C.defined() ? C.data_ptr<float>() : nullptr, a_inds.data_ptr<long>(),
          d_inds.data_ptr<long>(), D.data_ptr<float>(), M, K, N, P, Q);
    } else if (method == WMMA_F16_ACC_F32_DB_AMPERE) {
      implicit_gemm_wmma_f16_acc_f32_db_ampere<<<grid, dim3(32, 1, 1)>>>(
          reinterpret_cast<half *>(A.data_ptr<at::Half>()),
          reinterpret_cast<half *>(B.data_ptr<at::Half>()),
          C.defined() ? reinterpret_cast<half *>(C.data_ptr<at::Half>())
                      : nullptr,
          a_inds.data_ptr<long>(), d_inds.data_ptr<long>(), D.data_ptr<float>(),
          M, K, N, P, Q);
    } else if (method == WMMA_DB_AMPERE_GENERIC ||
               method == WMMA_DB_AMPERE_GENERIC_STORE) {
      if (method == WMMA_DB_AMPERE_GENERIC_STORE)
        use_store = true;
      auto Adtype = A.scalar_type();
      auto Ddtype = D.scalar_type();
      if (Adtype == at::kHalf && Ddtype == at::kFloat) {
        if (use_store) {
          launch_wmma_db_ampere_f16_f32_store(
              grid, dim3(32, 1, 1), nullptr,
              reinterpret_cast<half *>(A.data_ptr<at::Half>()),
              reinterpret_cast<half *>(B.data_ptr<at::Half>()),
              C.defined() ? reinterpret_cast<half *>(C.data_ptr<at::Half>())
                          : nullptr,
              a_inds.data_ptr<long>(), d_inds.data_ptr<long>(),
              D.data_ptr<float>(), M, K, N, P, Q);
        } else {
          launch_wmma_db_ampere_f16_f32(
              grid, dim3(32, 1, 1), nullptr,
              reinterpret_cast<half *>(A.data_ptr<at::Half>()),
              reinterpret_cast<half *>(B.data_ptr<at::Half>()),
              C.defined() ? reinterpret_cast<half *>(C.data_ptr<at::Half>())
                          : nullptr,
              a_inds.data_ptr<long>(), d_inds.data_ptr<long>(),
              D.data_ptr<float>(), M, K, N, P, Q);
        }
      } else if (Adtype == at::kHalf && Ddtype == at::kHalf) {
        if (use_store) {
          launch_wmma_db_ampere_f16_f16_store(
              grid, dim3(32, 1, 1), nullptr,
              reinterpret_cast<half *>(A.data_ptr<at::Half>()),
              reinterpret_cast<half *>(B.data_ptr<at::Half>()),
              C.defined() ? reinterpret_cast<half *>(C.data_ptr<at::Half>())
                          : nullptr,
              a_inds.data_ptr<long>(), d_inds.data_ptr<long>(),
              reinterpret_cast<half *>(D.data_ptr<at::Half>()), M, K, N, P, Q);
        } else {
          launch_wmma_db_ampere_f16_f16(
              grid, dim3(32, 1, 1), nullptr,
              reinterpret_cast<half *>(A.data_ptr<at::Half>()),
              reinterpret_cast<half *>(B.data_ptr<at::Half>()),
              C.defined() ? reinterpret_cast<half *>(C.data_ptr<at::Half>())
                          : nullptr,
              a_inds.data_ptr<long>(), d_inds.data_ptr<long>(),
              reinterpret_cast<half *>(D.data_ptr<at::Half>()), M, K, N, P, Q);
        }
      }
    } else if (method == CUB_F32_BLOCKLOAD) {
      auto Adtype = A.scalar_type();
      auto Bdtype = B.scalar_type();
      auto Ddtype = D.scalar_type();
      if (Adtype == at::kHalf && Bdtype == at::kHalf) {
        if (Ddtype == at::kFloat) {
          implicit_gemm_cub_blockload<half, half, half, float, 128, 4>
              <<<grid, dim3(128, 1, 1)>>>(
                  reinterpret_cast<half *>(A.data_ptr<at::Half>()),
                  reinterpret_cast<half *>(B.data_ptr<at::Half>()),
                  C.defined() ? reinterpret_cast<half *>(C.data_ptr<at::Half>())
                              : nullptr,
                  a_inds.data_ptr<long>(), d_inds.data_ptr<long>(),
                  D.data_ptr<float>(), M, K, N, P, Q);
        } else if (Ddtype == at::kHalf) {
          implicit_gemm_cub_blockload<half, half, half, half, 128, 4>
              <<<grid, dim3(128, 1, 1)>>>(
                  reinterpret_cast<half *>(A.data_ptr<at::Half>()),
                  reinterpret_cast<half *>(B.data_ptr<at::Half>()),
                  C.defined() ? reinterpret_cast<half *>(C.data_ptr<at::Half>())
                              : nullptr,
                  a_inds.data_ptr<long>(), d_inds.data_ptr<long>(),
                  reinterpret_cast<half *>(D.data_ptr<at::Half>()), M, K, N, P,
                  Q);
        } else if (Ddtype == at::kBFloat16) {
          implicit_gemm_cub_blockload<half, half, half, __hip_bfloat16, 128, 4>
              <<<grid, dim3(128, 1, 1)>>>(
                  reinterpret_cast<half *>(A.data_ptr<at::Half>()),
                  reinterpret_cast<half *>(B.data_ptr<at::Half>()),
                  C.defined() ? reinterpret_cast<half *>(C.data_ptr<at::Half>())
                              : nullptr,
                  a_inds.data_ptr<long>(), d_inds.data_ptr<long>(),
                  reinterpret_cast<__hip_bfloat16 *>(D.data_ptr<at::BFloat16>()),
                  M, K, N, P, Q);
        }
      } else if (Adtype == at::kBFloat16 && Bdtype == at::kBFloat16) {
        if (Ddtype == at::kFloat) {
          implicit_gemm_cub_blockload<__hip_bfloat16, __hip_bfloat16,
                                      __hip_bfloat16, float, 128, 4>
              <<<grid, dim3(128, 1, 1)>>>(
                  reinterpret_cast<__hip_bfloat16 *>(A.data_ptr<at::BFloat16>()),
                  reinterpret_cast<__hip_bfloat16 *>(B.data_ptr<at::BFloat16>()),
                  C.defined() ? reinterpret_cast<__hip_bfloat16 *>(
                                    C.data_ptr<at::BFloat16>())
                              : nullptr,
                  a_inds.data_ptr<long>(), d_inds.data_ptr<long>(),
                  D.data_ptr<float>(), M, K, N, P, Q);
        } else if (Ddtype == at::kBFloat16) {
          implicit_gemm_cub_blockload<__hip_bfloat16, __hip_bfloat16,
                                      __hip_bfloat16, __hip_bfloat16, 128, 4>
              <<<grid, dim3(128, 1, 1)>>>(
                  reinterpret_cast<__hip_bfloat16 *>(A.data_ptr<at::BFloat16>()),
                  reinterpret_cast<__hip_bfloat16 *>(B.data_ptr<at::BFloat16>()),
                  C.defined() ? reinterpret_cast<__hip_bfloat16 *>(
                                    C.data_ptr<at::BFloat16>())
                              : nullptr,
                  a_inds.data_ptr<long>(), d_inds.data_ptr<long>(),
                  reinterpret_cast<__hip_bfloat16 *>(D.data_ptr<at::BFloat16>()),
                  M, K, N, P, Q);
        } else if (Ddtype == at::kHalf) {
          implicit_gemm_cub_blockload<__hip_bfloat16, __hip_bfloat16,
                                      __hip_bfloat16, half, 128, 4>
              <<<grid, dim3(128, 1, 1)>>>(
                  reinterpret_cast<__hip_bfloat16 *>(A.data_ptr<at::BFloat16>()),
                  reinterpret_cast<__hip_bfloat16 *>(B.data_ptr<at::BFloat16>()),
                  C.defined() ? reinterpret_cast<__hip_bfloat16 *>(
                                    C.data_ptr<at::BFloat16>())
                              : nullptr,
                  a_inds.data_ptr<long>(), d_inds.data_ptr<long>(),
                  reinterpret_cast<half *>(D.data_ptr<at::Half>()), M, K, N, P,
                  Q);
        }
      }
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    times.push_back(ms);
  }
  hipEventDestroy(start);
  hipEventDestroy(stop);
  return times;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("benchmark_implicit_gemm", &benchmark_implicit_gemm,
        "Benchmark implicit GEMM: D[d]=A[a]@B + C[a]", py::arg("A"),
        py::arg("B"), py::arg("C"), py::arg("a_inds"), py::arg("d_inds"),
        py::arg("D"), py::arg("method"), py::arg("iterations") = 100,
        py::arg("use_store") = false);
}
